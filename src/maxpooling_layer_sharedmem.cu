#include "hip/hip_runtime.h"
//
// File:        maxpooling_layer.c
// Description: Implementation of max pooling layer
// Author:      Haris Wang
//
#include <stdlib.h>
#include <math.h>
#include <pthread.h>
#include <string.h>
#include "maxpooling_layer.h"
#include <hip/hip_runtime.h>
#include <float.h>

#define MAX(a, b) (((a) > (b)) ? (a) : (b))
#define MIN(a, b) (((a) < (b)) ? (a) : (b))

typedef struct mp_args
{
    max_pooling_op *op;
    short batch_id;
} mp_args;

#define WARP 32
#define THREADS_PER_BLOCK 512
#define WARPS_PER_BLOCK (THREADS_PER_BLOCK / WARP) // 16
#define TILE 16

// P_S: stride
// P_K: kernel size
__global__ void maxpool_forward_naive(const float *__restrict__ x,
                                      float *__restrict__ y,
                                      int N, int C, int H, int W,
                                      int OH, int OW, int P_S, int P_K)
{
    const int n = blockIdx.y;          // image id
    const int cg = blockIdx.x;         // 16-channel group
    const int warp = threadIdx.x >> 5; // 0..3
    const int lane = threadIdx.x & 31; // 0..31
    const int c = cg * WARPS_PER_BLOCK + warp;
    if (c >= C)
        return;

    // 4×8 warp tile coordinates
    const int WARP_H = 4, WARP_W = 8;
    const int local_h = lane / WARP_W; // 0..3
    const int local_w = lane % WARP_W; // 0..7
    const int tiles_h = (OH + WARP_H - 1) / WARP_H;
    const int tiles_w = (OW + WARP_W - 1) / WARP_W;

    const int SH_W = TILE + P_K - 1; // 10 when TILE = 8 and K = 3
    const int SH_H = SH_W;           // square
    const int SH_SIZE = SH_W * SH_H; // 100 floats
    extern __shared__ float sm[];    // shared memory for one Block
    float *warp_smem = sm + warp * SH_SIZE;

    for (int th = 0; th < tiles_h; ++th)
    {
        int oy0 = th * WARP_H; // top-left output y of tile
        int ih0 = oy0 * P_S;   // top-left input y

        for (int tw = 0; tw < tiles_w; ++tw)
        {
            int ox0 = tw * WARP_W;
            int iw0 = ox0 * P_S;

            // shared memory is not large enough to hold one image's single channel
            // so we need to load 10×10 tile (with halo) for each warp
            /* ---------- 1. load 10×10 tile (+halo) ---------- */
            // if condition: we only need certain threads in the warp to load
            // the shareed memory, this is deceided by the tile size, the kernel size and the stride
            if (((SH_W - P_K + 1) / P_S == 0) || lane % ((SH_W - P_K + 1) / P_S) == 0)
            {
                // load the first row
                for (int t = lane; t < SH_W; t += WARP)
                {
                    int dy = t / SH_W; // 0..9
                    int dx = t % SH_W; // 0..9
                    int ih = ih0 + dy;
                    int iw = iw0 + dx;

                    float v = (ih < H && iw < W)
                                  ? x[((n * C + c) * H + ih) * W + iw]
                                  : -FLT_MAX;
                    warp_smem[dy * SH_W + dx] = v;
                }
            }
            __syncwarp(); // tile ready

            /* ---------- 2. compute this warp’s 4×4 outputs ---------- */
            int oy = oy0 + local_h;
            int ox = ox0 + local_w;
            if (oy < OH && ox < OW)
            {
                float vmax = -FLT_MAX;
                for (int ky = 0; ky < P_K; ++ky)
                    for (int kx = 0; kx < P_K; ++kx)
                    {
                        float v = warp_smem[(local_h * P_S + ky) * SH_W + (local_w * P_S + kx)];
                        vmax = fmaxf(vmax, v);
                    }
                y[((n * C + c) * OH + oy) * OW + ox] = vmax;
            }
            __syncwarp(); // allow sm reuse
        }
    }
}

void max_pooling_op_forward(max_pooling_op *op)
{
    int N = op->batchsize;
    int C = op->channels;
    int H = op->in_h;
    int W = op->in_w;
    int OH = op->out_h;
    int OW = op->out_w;

    size_t in_bytes = (size_t)N * C * H * W * sizeof(float);
    size_t out_bytes = (size_t)N * C * OH * OW * sizeof(float);

    /* ---------- allocate ---------- */
    float *d_in, *d_out;
    hipMalloc(&d_in, in_bytes);
    hipMalloc(&d_out, out_bytes);

    /* ---------- H→D copy ---------- */
    hipMemcpy(d_in, op->input, in_bytes, hipMemcpyHostToDevice);

    constexpr int TPB = THREADS_PER_BLOCK;
    int num_c_blocks = (C + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
    dim3 blockDim(TPB);
    dim3 gridDim(num_c_blocks, N);

    /* ---------- kernel launch ---------- */
    size_t smem_bytes = WARPS_PER_BLOCK * (TILE + op->kernel_size - 1) * (TILE + op->kernel_size - 1) * sizeof(float);
    maxpool_forward_naive<<<gridDim, blockDim, smem_bytes>>>(d_in, d_out, N, C, H, W, OH, OW, op->stride, op->kernel_size);
    hipDeviceSynchronize();
    /* ---------- D→H copy ---------- */
    hipMemcpy(op->output, d_out, out_bytes, hipMemcpyDeviceToHost);
    /* ---------- free memory ---------- */
    hipFree(d_in);
    hipFree(d_out);
}

void max_pooling_op_backward(max_pooling_op *op)
{
    int channels = op->channels;
    int pool_size = op->kernel_size;
    int in_w = op->in_w;
    int in_h = op->in_h;
    int out_w = op->out_w;
    int out_h = op->out_h;
    register int iwih = in_w * in_h;
    register int owoh = out_w * out_h;

    int in_x, in_y;
    float max_value, cur_value;
    int x, y;
    register int in_shift, out_shift;
    for (int c = 0; c < channels; c++)
    {
        for (int i = 0; i < op->out_w; i++)
        {
            for (int j = 0; j < op->out_h; j++)
            {
                for (int p = 0; p < op->batchsize; p++)
                {
                    //
                    // output[p][c][i][j]
                    //
                    x = i * pool_size;
                    y = j * pool_size;
                    max_value = -1111111;
                    while (x < MIN((i + 1) * pool_size, in_w))
                    {
                        while (y < MIN((j + 1) * pool_size, in_h))
                        {
                            cur_value = op->input[p * channels * iwih + c * iwih + y * in_w + x];
                            if (cur_value > max_value)
                            {
                                max_value = cur_value;
                                in_x = x;
                                in_y = y;
                            }
                            y++;
                        }
                        x++;
                    }

                    in_shift = c * iwih + in_y * in_w + in_x;
                    out_shift = c * owoh + j * out_w + i;
                    op->d_input[in_shift] += op->d_output[out_shift] / op->batchsize;
                }
            }
        }
    }
}