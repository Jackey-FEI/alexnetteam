#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <pthread.h>
#include <string.h>
#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "convolution_layer.h"

__global__ void conv2d_forward_kernel(
    const float *input,
    const float *weights,
    const float *bias,
    float *output,
    int batchsize,
    int in_c, int in_h, int in_w,
    int out_c, int out_h, int out_w,
    int ksize, int stride,
    int in_units, int out_units)
{
    int batch_id = blockIdx.z;
    int oc = blockIdx.y;
    int oh = blockIdx.x / out_w;
    int ow = blockIdx.x % out_w;

    int output_idx = batch_id * out_units + oc * (out_h * out_w) + oh * out_w + ow;
    float sum = bias[oc];

    for (int ic = 0; ic < in_c; ++ic) {
        for (int kh = 0; kh < ksize; ++kh) {
            for (int kw = 0; kw < ksize; ++kw) {
                int ih = oh * stride + kh;
                int iw = ow * stride + kw;

                if (ih < in_h && iw < in_w) {
                    int input_idx = batch_id * in_units + ic * in_h * in_w + ih * in_w + iw;
                    int weight_idx = oc * (in_c * ksize * ksize) + ic * ksize * ksize + kh * ksize + kw;

                    sum += input[input_idx] * weights[weight_idx];
                }
            }
        }
    }

    output[output_idx] = sum;
}

static void img2col(const float *img, float *col, const conv_op *op)
{
    /**
     * Output
     *      col[ikk][owoh]
     * */

    //
    // Todo: simplify the code
    //
    register int input_offset;
    register int iwih = op->in_w*op->in_h;
    register int kk   = op->kernel_size* op->kernel_size;
    register int ikk  = op->in_channels * kk;
    register const float *input = img;
    register float *x_col = col;
    for (register unsigned short in_c = 0; in_c < op->in_channels; in_c++)
    {
        register int x_col_offset = in_c * kk;
        for (register int st_x = 0; st_x < op->out_w * op->stride; st_x += op->stride)
        {
            for (register int st_y = 0; st_y < op->out_h * op->stride; st_y += op->stride, x_col_offset += ikk)
            {
                for (register unsigned short j = 0; j < op->kernel_size; j++)
                {
                    for (register unsigned short i = 0; i < op->kernel_size; i++, x_col_offset++)
                    {
                        if (!(st_x+i <op->in_w) | !(st_y+j <op->in_h))
                        {
                            x_col[x_col_offset] = 0;
                            continue;
                        }

                        input_offset = (st_x+i) + (st_y+j) * op->in_w + in_c * iwih;
                        x_col[x_col_offset] = input[input_offset];
                    }
                }
            }
        }
        ikk += kk;
    }
}

__host__ void conv_op_forward(conv_op *op) {
    op->input_col = (float *)calloc((op->batchsize)*(op->in_channels * op->kernel_size* op->kernel_size)*(op->out_w * op->out_h), sizeof(float));
    for(int p = 0; p < op->batchsize; p++) {
        float *x_col    = op->input_col + p * op->in_units;
        float *t_input  = op->input + p * op->in_units;
        img2col(t_input, x_col, op);
    }

    int in_units = op->in_channels * op->in_h * op->in_w;
    int out_units = op->out_channels * op->out_h * op->out_w;

    float *d_input, *d_weights, *d_bias, *d_output;

    size_t input_size = sizeof(float) * op->batchsize * in_units;
    size_t weight_size = sizeof(float) * op->out_channels * op->in_channels * op->kernel_size * op->kernel_size;
    size_t bias_size = sizeof(float) * op->out_channels;
    size_t output_size = sizeof(float) * op->batchsize * out_units;

    // Allocate device memory
    hipMalloc(&d_input, input_size);
    hipMalloc(&d_weights, weight_size);
    hipMalloc(&d_bias, bias_size);
    hipMalloc(&d_output, output_size);

    // Copy data to device
    hipMemcpy(d_input, op->input, input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_weights, op->weights, weight_size, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, op->bias, bias_size, hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    // Configure grid and block dimensions
    dim3 blockDim(1);
    dim3 gridDim(op->out_w * op->out_h, op->out_channels, op->batchsize);

    // Launch the kernel
    conv2d_forward_kernel<<<gridDim, blockDim>>>(
        d_input, d_weights, d_bias, d_output,
        op->batchsize,
        op->in_channels, op->in_h, op->in_w,
        op->out_channels, op->out_h, op->out_w,
        op->kernel_size, op->stride,
        in_units, out_units
    );

    // Copy the result back to host
    hipMemcpy(op->output, d_output, output_size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    // Free device memory
    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_bias);
    hipFree(d_output);
}
