#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <pthread.h>
#include <string.h>
#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "convolution_layer.h"

#define threads_per_block 512
#define warps_per_block (threads_per_block / 32) // 16

__global__ void conv2d_forward_kernel(
    const float *input,
    const float *weights,
    const float *bias,
    float *output,
    int batchsize,
    int in_c, int in_h, int in_w,
    int out_c, int out_h, int out_w,
    int ksize, int stride,
    int in_units, int out_units)
{
    int batch_id = blockIdx.y; // range[0 ~ batchsize)
    int channel_block_id = blockIdx.x; // range[0 ~ num_channel_blocks)
    int tid = threadIdx.x; // range[0 ~ threads_per_block 512)
    int warp_id = tid / 32; // range[0 ~ warps_per_block 16)
    int lane_id = tid % 32; // range[0 ~ 32)
    int oc = channel_block_id * warps_per_block + warp_id; // range[0 ~ out_c)

    if (oc >= out_c) return;

    __syncthreads();

    int lane_cnt = lane_id;
    while(lane_cnt < out_h * out_w) {
        // obtain oh and ow based on the lane_cnt (position of each thread in one image)
        int oh = lane_cnt / out_w;
        int ow = lane_cnt % out_w;
        float sum = bias[oc];
        // loop through all input channels
        for (int ic = 0; ic < in_c; ++ic) {
            // convolution
            for (int kh = 0; kh < ksize; ++kh) {
                for (int kw = 0; kw < ksize; ++kw) {
                    // obtain ih and iw
                    int ih = oh * stride + kh;
                    int iw = ow * stride + kw;

                    if (ih < in_h && iw < in_w) {
                        int input_idx = batch_id * in_units + ic * in_h * in_w + ih * in_w + iw;
                        int weight_idx = oc * in_c * ksize * ksize + ic * ksize * ksize + kh * ksize + kw;
                        sum += input[input_idx] * weights[weight_idx];
                    }
                }
            }
        }
        int output_idx = batch_id * out_units + oc * out_h * out_w + oh * out_w + ow;
        output[output_idx] = sum;

        // warp move to next computation
        lane_cnt += 32;
    }
}

static void img2col(const float *img, float *col, const conv_op *op)
{
    /**
     * Output
     *      col[ikk][owoh]
     * */

    //
    // Todo: simplify the code
    //
    register int input_offset;
    register int iwih = op->in_w*op->in_h;
    register int kk   = op->kernel_size* op->kernel_size;
    register int ikk  = op->in_channels * kk;
    register const float *input = img;
    register float *x_col = col;
    for (register unsigned short in_c = 0; in_c < op->in_channels; in_c++)
    {
        register int x_col_offset = in_c * kk;
        for (register int st_x = 0; st_x < op->out_w * op->stride; st_x += op->stride)
        {
            for (register int st_y = 0; st_y < op->out_h * op->stride; st_y += op->stride, x_col_offset += ikk)
            {
                for (register unsigned short j = 0; j < op->kernel_size; j++)
                {
                    for (register unsigned short i = 0; i < op->kernel_size; i++, x_col_offset++)
                    {
                        if (!(st_x+i <op->in_w) | !(st_y+j <op->in_h))
                        {
                            x_col[x_col_offset] = 0;
                            continue;
                        }

                        input_offset = (st_x+i) + (st_y+j) * op->in_w + in_c * iwih;
                        x_col[x_col_offset] = input[input_offset];
                    }
                }
            }
        }
        ikk += kk;
    }
}

static void print_conv_op(conv_op *op) {
    printf(">>>>>>>>>>>>>>>>> conv >>>>>>>>>>>>>>>>>>>\n");
    printf("in channels: %d \n", op->in_channels);
    printf("out channels: %d \n", op->out_channels);
    printf("kernel size: %d \n", op->kernel_size);
    printf("padding: %d \n", op->padding);
    printf("stride: %d \n", op->stride);
    printf("in width: %d \n", op->in_w);
    printf("in height: %d \n", op->in_h);
    printf("out width: %d \n", op->out_w);
    printf("out height: %d \n", op->out_h);
    printf("in units: %d \n", op->in_units);
    printf("out units: %d \n", op->out_units);
    printf("batch size: %d \n", op->batchsize);
    printf(">>>>>>>>>>>>>>>>>> conv >>>>>>>>>>>>>>>>>>\n");
}

// void nchw_to_rowmajor(float* dst, const float* src, int batch, int c, int h, int w) {
//     int hw = h * w;
//     int chw = c * hw;
//     for (int n = 0; n < batch; ++n) {
//         for (int h_i = 0; h_i < h; ++h_i) {
//             for (int w_i = 0; w_i < w; ++w_i) {
//                 for (int c_i = 0; c_i < c; ++c_i) {
//                     int rowmajor_idx = n * chw + h_i * w * c + w_i * c + c_i;
//                     int nchw_idx = n * chw + c_i * hw + h_i * w + w_i;
//                     dst[rowmajor_idx] = src[nchw_idx];
//                 }
//             }
//         }
//     }
// }

// void rowmajor_to_nchw(float* dst, const float* src, int batch, int c, int h, int w) {
//     int hw = h * w;
//     int chw = c * hw;
//     for (int n = 0; n < batch; ++n) {
//         for (int h_i = 0; h_i < h; ++h_i) {
//             for (int w_i = 0; w_i < w; ++w_i) {
//                 for (int c_i = 0; c_i < c; ++c_i) {
//                     int rowmajor_idx = n * chw + h_i * w * c + w_i * c + c_i;
//                     int nchw_idx = n * chw + c_i * hw + h_i * w + w_i;
//                     dst[nchw_idx] = src[rowmajor_idx];
//                 }
//             }
//         }
//     }
// }

__host__ void conv_op_forward(conv_op *op) {
    /* NOTE: allocate for backward */
    op->input_col = (float *)calloc((op->batchsize+1)*(op->in_channels * op->kernel_size* op->kernel_size)*(op->out_w * op->out_h), sizeof(float));
    for(int p = 0; p < op->batchsize; p++) {
        float *x_col    = op->input_col + p * op->in_units;
        float *t_input  = op->input + p * op->in_units;
        img2col(t_input, x_col, op);
    }
    /* NOTE: do not remove */

    int in_units = op->in_channels * op->in_h * op->in_w;
    int out_units = op->out_channels * op->out_h * op->out_w;

    // // Convert input and weight from row-major to channel-major (NCHW)
    // float *converted_input = (float *)malloc(sizeof(float) * op->batchsize * in_units);
    // rowmajor_to_nchw(converted_input, op->input, op->batchsize, op->in_channels, op->in_h, op->in_w);

    float *d_input, *d_weights, *d_bias, *d_output;
    size_t input_size = sizeof(float) * op->batchsize * in_units;
    size_t weight_size = sizeof(float) * op->out_channels * op->in_channels * op->kernel_size * op->kernel_size;
    size_t bias_size = sizeof(float) * op->out_channels;
    size_t output_size = sizeof(float) * op->batchsize * out_units;
    // float *converted_weights = (float *)malloc(weight_size);
    // rowmajor_to_nchw(converted_weights, op->weights, op->out_channels, op->in_channels, op->kernel_size, op->kernel_size);

    // Cuda memory malloc
    hipMalloc(&d_input, input_size);
    hipMalloc(&d_weights, weight_size);
    hipMalloc(&d_bias, bias_size);
    hipMalloc(&d_output, output_size);

    // hipMemcpy(d_input, converted_input, input_size, hipMemcpyHostToDevice);
    // hipMemcpy(d_weights, converted_weights, weight_size, hipMemcpyHostToDevice);
    hipMemcpy(d_input, op->input, input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_weights, op->weights, weight_size, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, op->bias, bias_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // Thread parameters
    const int num_channel_blocks = (op->out_channels + warps_per_block - 1) / warps_per_block;

    dim3 blockDim(threads_per_block);
    dim3 gridDim(num_channel_blocks, op->batchsize);

    conv2d_forward_kernel<<<gridDim, blockDim>>>(
        d_input, d_weights, d_bias, d_output,
        op->batchsize,
        op->in_channels, op->in_h, op->in_w,
        op->out_channels, op->out_h, op->out_w,
        op->kernel_size, op->stride,
        in_units, out_units
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Kernel execution failed: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(op->output, d_output, output_size, hipMemcpyDeviceToHost);
    // Convert output from channel-major (NCHW) to row-major
    // float *converted_output = (float *)malloc(sizeof(float) * op->batchsize * out_units);
    // hipMemcpy(converted_output, d_output, output_size, hipMemcpyDeviceToHost);
    // nchw_to_rowmajor(op->output, converted_output, op->batchsize, op->out_channels, op->out_h, op->out_w);

    // Clean up
    // free(converted_input);
    // free(converted_weights);
    // free(converted_output);
    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_bias);
    hipFree(d_output);
}

